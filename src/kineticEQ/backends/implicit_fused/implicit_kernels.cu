#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>
#include <assert.h>

namespace implicit_fused {

// ================= Utility =================
template <typename T>
__device__ __forceinline__ T clamp_pos(T x, T eps) {
    return (x > eps) ? x : eps;
}

// Maxwellian for one (n,u,T) row and one v
template <typename T>
__device__ __forceinline__ T maxwell_1v(T n, T u, T Tgas, T vj, T inv_sqrt_2pi) {
    // fM = n / sqrt(2π T) * exp(-(v-u)^2/(2T))
    T inv_sqrtT = rsqrt(Tgas);
    T coeff = n * inv_sqrt_2pi * inv_sqrtT;
    T diff  = vj - u;
    T expo  = -T(0.5) * (diff*diff) / Tgas;
    return coeff * exp(expo);
}

// ================= Moments kernel =================
// 1 block per row (x), reduction in shared memory.
template <typename T>
__global__ void moments_kernel(
    const T* __restrict__ f,
    const T* __restrict__ v,
    int nx, int nv, T dv,
    T* __restrict__ n_out,
    T* __restrict__ u_out,
    T* __restrict__ T_out)
{
    const int i = blockIdx.x;
    if (i >= nx) return;

    extern __shared__ unsigned char smem_raw[];
    T* s0 = reinterpret_cast<T*>(smem_raw);
    T* s1 = reinterpret_cast<T*>(smem_raw + sizeof(T)*blockDim.x);
    T* s2 = reinterpret_cast<T*>(smem_raw + sizeof(T)*blockDim.x*2);

    T p0 = T(0), p1 = T(0), p2 = T(0);
    for (int j = threadIdx.x; j < nv; j += blockDim.x) {
        T fij = f[i*nv + j];
        T vj  = v[j];
        p0 += fij;
        p1 += fij * vj;
        p2 += fij * vj * vj;
    }
    s0[threadIdx.x] = p0;
    s1[threadIdx.x] = p1;
    s2[threadIdx.x] = p2;
    __syncthreads();

    for (int offset = blockDim.x>>1; offset>0; offset >>= 1) {
        if (threadIdx.x < offset) {
            s0[threadIdx.x] += s0[threadIdx.x + offset];
            s1[threadIdx.x] += s1[threadIdx.x + offset];
            s2[threadIdx.x] += s2[threadIdx.x + offset];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        T n  = s0[0] * dv;
        T s1d= s1[0] * dv;
        T s2d= s2[0] * dv;
        T u  = s1d / n;
        T Tg = s2d / n - u*u;
        if (!(Tg > T(0))) Tg = T(1e-300);
        n_out[i] = n;
        u_out[i] = u;
        T_out[i] = Tg;
    }
}

// ================= Build tri-diagonal & RHS =================
// 1 block per velocity j. Build dl/d/du/B for interior cells k=0..n_inner-1 (i=k+1)
template <typename T>
__global__ void build_tridiag_rhs_kernel(
    const T* __restrict__ f,
    const T* __restrict__ v,
    const T* __restrict__ n,
    const T* __restrict__ u,
    const T* __restrict__ Tg,
    const T* __restrict__ fL,
    const T* __restrict__ fR,
    int nx, int nv, T dt, T dx, T tau_tilde, T inv_sqrt_2pi,
    T* __restrict__ dl,  // (nv, n_inner)
    T* __restrict__ dd,  // (nv, n_inner)
    T* __restrict__ du,  // (nv, n_inner)
    T* __restrict__ B)   // (nv, n_inner)
{
    const int j = blockIdx.x;
    if (j >= nv) return;
    const int n_inner = nx - 2;
    const T vj = v[j];
    const T ap = -dt/dx * fmax(vj, T(0));   // a_coeff (<=0)
    const T cp = -dt/dx * fmax(-vj, T(0));  // c_coeff (<=0)
    const T minus_a = -ap; // = dt/dx*max(v,0)
    const T minus_c = -cp; // = dt/dx*max(-v,0)

    T* dl_j = dl + j * n_inner;
    T* dd_j = dd + j * n_inner;
    T* du_j = du + j * n_inner;
    T*  B_j =  B + j * n_inner;

    // boundary Maxwellians for this velocity
    const T fL_j = fL[j];
    const T fR_j = fR[j];

    for (int k = threadIdx.x; k < n_inner; k += blockDim.x) {
        const int i = k + 1; // interior row index in [1..nx-2]

        // inv_tau = (n*sqrt(T))/tau_tilde
        const T sqrtT = sqrt(Tg[i]);
        const T inv_tau = (n[i] * sqrtT) / tau_tilde;

        // diagonal
        dd_j[k] = T(1) + minus_a + minus_c + dt * inv_tau;
        // sub & super
        dl_j[k] = (k==0) ? T(0) : ap;
        du_j[k] = (k==n_inner-1) ? T(0) : cp;

        // Maxwellian at interior cell
        const T fM = maxwell_1v<T>(n[i], u[i], Tg[i], vj, inv_sqrt_2pi);

        // RHS
        const T fij = f[i*nv + j];
        T rhs = fij + dt * inv_tau * fM;

        // boundary advection contributions
        if (k == 0)        rhs += (dt/dx) * fmax(vj,  T(0)) * fL_j;
        if (k == n_inner-1)rhs += (dt/dx) * fmax(-vj, T(0)) * fR_j;

        B_j[k] = rhs;
    }
}

// ================= Launchers =================
void launch_moments_double(
    const double* f, const double* v,
    int nx, int nv, double dv,
    double* n, double* u, double* T,
    hipStream_t stream)
{
    const int block = 256;
    dim3 grid(nx);
    size_t shmem = sizeof(double) * block * 3;
    moments_kernel<double><<<grid, block, shmem, stream>>>(
        f, v, nx, nv, dv, n, u, T);
}

void launch_build_tridiag_rhs_double(
    const double* f, const double* v,
    const double* n, const double* u, const double* T,
    const double* fL, const double* fR,
    int nx, int nv, double dt, double dx, double tau_tilde, double inv_sqrt_2pi,
    double* dl, double* d, double* du, double* B,
    hipStream_t stream)
{
    const int block = 256;
    dim3 grid(nv);
    build_tridiag_rhs_kernel<double><<<grid, block, 0, stream>>>(
        f, v, n, u, T, fL, fR, nx, nv, dt, dx, tau_tilde, inv_sqrt_2pi, dl, d, du, B);
}

template <typename T>
__global__ void boundary_maxwell_kernel(
    const T* __restrict__ v, int nv, T inv_sqrt_2pi,
    T nL, T uL, T TL, T nR, T uR, T TR,
    T* __restrict__ fL, T* __restrict__ fR)
{
    TL = clamp_pos(TL, T(1e-300));
    TR = clamp_pos(TR, T(1e-300));
    T inv_sqrtTL = rsqrt(TL);
    T inv_sqrtTR = rsqrt(TR);
    T coeffL = nL * inv_sqrt_2pi * inv_sqrtTL;
    T coeffR = nR * inv_sqrt_2pi * inv_sqrtTR;
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < nv; j += blockDim.x * gridDim.x) {
        T vj = v[j];
        T eL = exp(-T(0.5) * (vj-uL)*(vj-uL) / TL);
        T eR = exp(-T(0.5) * (vj-uR)*(vj-uR) / TR);
        fL[j] = coeffL * eL;
        fR[j] = coeffR * eR;
    }
}

void launch_boundary_maxwell_double(
    const double* v, int nv, double inv_sqrt_2pi,
    double nL, double uL, double TL,
    double nR, double uR, double TR,
    double* fL, double* fR,
    hipStream_t stream)
{
    int block = 256;
    int grid = (nv + block - 1) / block;
    boundary_maxwell_kernel<double><<<grid, block, 0, stream>>>(
        v, nv, inv_sqrt_2pi, nL, uL, TL, nR, uR, TR, fL, fR);
}

} // namespace implicit_fused
